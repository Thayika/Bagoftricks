#include "hip/hip_runtime.h"
#include <iostream>
#include <mutex>
#include "cudaloss.h"
#include "matrix.h"
#include "densematrix.h"
#include "vector.h"

namespace fasttext {
static const float one = 1.0;
static const float zero = 0.0;
static const real epsilon = 0.00001f;
real* CudaSoftmaxLoss::d_wo_;

#define CUDA_CHECK(error) { \
  if (error!=hipSuccess){ \
    std::cerr<<"CUDA ERROR "<< hipGetErrorName(error) << " in file "  << __FILE__ << " line " <<__LINE__<< std::endl; \
    exit(0); \
  }  \
}

static const char* cublasGetErrorEnum(hipblasStatus_t error)
{
  switch (error)
  {
    case HIPBLAS_STATUS_SUCCESS:
      return "HIPBLAS_STATUS_SUCCESS";
    case HIPBLAS_STATUS_NOT_INITIALIZED:
      return "HIPBLAS_STATUS_NOT_INITIALIZED";
    case HIPBLAS_STATUS_ALLOC_FAILED:
      return "HIPBLAS_STATUS_ALLOC_FAILED";
    case HIPBLAS_STATUS_INVALID_VALUE:
      return "HIPBLAS_STATUS_INVALID_VALUE";
    case HIPBLAS_STATUS_ARCH_MISMATCH:
      return "HIPBLAS_STATUS_ARCH_MISMATCH";
    case HIPBLAS_STATUS_MAPPING_ERROR:
      return "HIPBLAS_STATUS_MAPPING_ERROR";
    case HIPBLAS_STATUS_EXECUTION_FAILED:
      return "HIPBLAS_STATUS_EXECUTION_FAILED";
    case HIPBLAS_STATUS_INTERNAL_ERROR:
      return "HIPBLAS_STATUS_INTERNAL_ERROR";
    case HIPBLAS_STATUS_NOT_SUPPORTED:
      return "HIPBLAS_STATUS_NOT_SUPPORTED";
    case HIPBLAS_STATUS_UNKNOWN:
      return "HIPBLAS_STATUS_UNKNOWN";
    default:
      return "UNKNOWN CUBLAS ERROR";
  }
}

#define CUBLAS_CHECK(err) \
{ \
  if (HIPBLAS_STATUS_SUCCESS != err) { \
    std::cerr<<"CUBLAS ERROR "<< cublasGetErrorEnum(err) << " in file "  << __FILE__ << " line " <<__LINE__<< std::endl; \
    exit(0); \
  } \
}

CudaState::CudaState(int32_t hiddenSize, int32_t outputSize, int32_t seed)
	:Model::State(hiddenSize, outputSize, seed), ifs_(NULL) {
  int64_t M = outputSize;
  int64_t N = hiddenSize;
  CUDA_CHECK(hipMalloc((void**)&d_hidden_, N*sizeof(real)));
  CUDA_CHECK(hipMalloc((void**)&d_output_, M*sizeof(real)));
  CUDA_CHECK(hipMalloc((void**)&d_softmax_output_, M*sizeof(real)));
  CUDA_CHECK(hipMalloc((void**)&d_output_diff_, M*sizeof(real)));
  CUDA_CHECK(hipMalloc((void**)&d_grad_, N*sizeof(real)));
  CUDA_CHECK(hipMalloc((void**)&d_lossValue_, sizeof(real)));

  stream_ = hipStreamPerThread;
  hipdnnCreate(&cudnn_);
  hipdnnCreateTensorDescriptor(&cudnn_output_desc_);
  hipdnnSetTensor4dDescriptor(cudnn_output_desc_, hipdnnTensorFormat_t::HIPDNN_TENSOR_NCHW, hipdnnDataType_t::HIPDNN_DATA_FLOAT, 1, 1, 1, M);
  hipdnnSetStream(cudnn_, stream_);
  CUBLAS_CHECK(hipblasCreate(&cublas_));
  CUBLAS_CHECK(hipblasSetStream(cublas_, stream_));
}

CudaState::~CudaState() {
  CUDA_CHECK(hipFree(d_hidden_));
  CUDA_CHECK(hipFree(d_output_));
  CUDA_CHECK(hipFree(d_softmax_output_));
  CUDA_CHECK(hipFree(d_output_diff_));
  CUDA_CHECK(hipFree(d_grad_));
  CUDA_CHECK(hipFree(d_lossValue_));
  hipdnnDestroyTensorDescriptor(cudnn_output_desc_);
  hipdnnDestroy(cudnn_);
  CUBLAS_CHECK(hipblasDestroy(cublas_));
}

int64_t CudaState::getLine(std::ifstream& ifs, std::shared_ptr<fasttext::Dictionary> dict, model_name model) {
  if( ifs_==NULL ) {
    ifs_ = &ifs;
    dict_ = dict;
    model_ = model;
    return State::getLine(ifs, dict, model);
  } else if( preLoadResult_==0 || labels.size()==0 || line.size()==0 ) {
    return State::getLine(ifs, dict, model);
  } else {
    return preLoadResult_;
  }
}

void CudaState::preLoadLine() {
  preLoadResult_ = State::getLine(*ifs_, dict_, model_);	
}

CudaSoftmaxLoss::CudaSoftmaxLoss(std::shared_ptr<Matrix>& wi, std::shared_ptr<Matrix>& wo):SoftmaxLoss(wo), wi_(wi) {
}

CudaSoftmaxLoss::~CudaSoftmaxLoss() {
}

bool CudaSoftmaxLoss::init() {
  // Copy wo from host to device
  int64_t m = wo_->size(0);
  int64_t n = wo_->size(1);
  std::vector<real> tmpwo(m*n);
  real* pBegin = tmpwo.data();
  for( int64_t i=0; i<m; i++ ) {
    Vector v(n);
    wo_->addRowToVector(v, i);
    memcpy(pBegin+i*n, v.data(), n*sizeof(real));
  }

  CUDA_CHECK(hipMalloc((void**)&d_wo_, m*n*sizeof(real)));
  CUDA_CHECK(hipMemcpy(d_wo_, pBegin, m*n*sizeof(real), hipMemcpyHostToDevice));
  return true;
}

void CudaSoftmaxLoss::shutdown() {
  // Copy wo from device back to host
  int64_t m = wo_->size(0);
  int64_t n = wo_->size(1);
  std::vector<real> tmpwo(m*n);
  CUDA_CHECK(hipMemcpy(tmpwo.data(), d_wo_, m*n*sizeof(real), hipMemcpyDeviceToHost));
  real* pBegin = tmpwo.data();
  for( int64_t i=0; i<m; i++ ) {
    Vector clear(n);
    wo_->addRowToVector(clear, i);
    wo_->addVectorToRow(clear, i, -1.0);

    Vector add(n);
    memcpy(add.data(), pBegin+i*n, n*sizeof(real));
    wo_->addVectorToRow(add, i, 1.0);
  }
  CUDA_CHECK(hipFree(d_wo_));
}

real CudaSoftmaxLoss::forward(
      const std::vector<int32_t>& targets,
      int32_t targetIndex,
      Model::State& state,
      real lr,
      bool backprop) {
  real gpuLoss = 0;
  CudaState& gpuState = (CudaState&)state;

#ifdef FASTTEXT_CUDA_DEBUG
  Model::State cpuState(state);
  compare(cpuState, gpuState, true, false);
  real cpuLoss = SoftmaxLoss::forward(targets, targetIndex, cpuState, lr, backprop);  
#endif

  cudaforward(gpuState, targets[targetIndex], lr, backprop, gpuLoss, gpuState.grad);
#ifdef FASTTEXT_CUDA_DEBUG
  compare(cpuState, gpuState, true, true);
  if( fabs(gpuLoss-cpuLoss)>epsilon )
    printf("Loss not match, cpu: %f, gpu: %f\n", cpuLoss, gpuLoss);
#endif

  return gpuLoss;
}

__global__
void CudacomputeDiff(real* softmax_output, size_t output_n, real* output_diff, real* loss, int32_t target, real lr) {
  int output_idx = blockIdx.x*blockDim.x + threadIdx.x;

  if( threadIdx.x==0 && blockIdx.x==0 ) {
    *loss = -std::log(softmax_output[target] + 1e-5);
  }

  if( output_idx < output_n ) {
    real label = (output_idx==target)?1.0:0.0;
    output_diff[output_idx] = lr * (label - softmax_output[output_idx]);
  }
}

void CudaSoftmaxLoss::compare(const Model::State& CPUState, const CudaState& GPUState, bool CmpWo, bool CmpSoftmaxOutput) {
  if( CmpWo ) {
    int64_t m = wo_->size(0);
    int64_t n = wo_->size(1);
    DenseMatrix* wo = dynamic_cast<DenseMatrix*>(wo_.get());
    std::vector<real> tmpwo(m*n);
    CUDA_CHECK(hipMemcpy(tmpwo.data(), d_wo_, m*n*sizeof(real), hipMemcpyDeviceToHost));
    for( int64_t i=0; i<m; i++ ) {
      for( int64_t j=0; j<n; j++ ) {
        if( fabs(tmpwo[i*n+j]-wo->at(i,j))>epsilon )
          printf("\nwo[%ld,%ld] not match %f %f\n", i, j, tmpwo[i*n+j], wo->at(i,j));
      }
    }
  }
  if( CmpSoftmaxOutput ) {
    int64_t m = wo_->size(0);
    std::vector<real> tmpSoftMax(m);
    CUDA_CHECK(hipMemcpy(tmpSoftMax.data(), GPUState.d_softmax_output_, m*sizeof(real), hipMemcpyDeviceToHost));
    for( int64_t i=0; i<m; i++ ) {
      if( fabs(tmpSoftMax[i]-CPUState.output[i])>epsilon )
	printf("\nsoftmax [%ld] not match %f %f\n", i, tmpSoftMax[i], CPUState.output[i]);
    }
  }
}

void CudaSoftmaxLoss::cudaforward(
      CudaState& batchState,
      int32_t target,
      real lr,
      bool backprop,
      real& lossValue,
      Vector& grad) {
  int M = wo_->size(0);  // labels
  int N = wo_->size(1);  // dims

  // Copy hidden from host to device
  CUDA_CHECK(hipMemcpyAsync(batchState.d_hidden_, batchState.hidden.data(), N*sizeof(real), hipMemcpyHostToDevice, batchState.stream_));

  // compute output
  CUBLAS_CHECK(hipblasSgemv(batchState.cublas_, HIPBLAS_OP_T,
    N, M,
    &one,
    d_wo_, N,
    batchState.d_hidden_, 1,
    &zero,
    batchState.d_output_, 1));

  // compute softmax
  hipdnnSoftmaxForward(batchState.cudnn_, hipdnnSoftmaxAlgorithm_t::HIPDNN_SOFTMAX_ACCURATE, hipdnnSoftmaxMode_t::HIPDNN_SOFTMAX_MODE_INSTANCE,
    &one, batchState.cudnn_output_desc_, batchState.d_output_,
    &zero, batchState.cudnn_output_desc_, batchState.d_softmax_output_);  

  // compute loss
  CudacomputeDiff<<<(M+255)/256, 256, 0, batchState.stream_>>>(
    batchState.d_softmax_output_,
    M,
    batchState.d_output_diff_,
    batchState.d_lossValue_,
    target, lr);

  if( backprop ) {
    // compute grad
    CUBLAS_CHECK(hipblasSgemv(batchState.cublas_, HIPBLAS_OP_T,
      M, N,
      &one,
      d_wo_, M,
      batchState.d_output_diff_, 1,
      &zero,
      batchState.d_grad_, 1));

    // update wo
    CUBLAS_CHECK(hipblasSger(batchState.cublas_,
      N, M,
      &one,
      batchState.d_hidden_, 1,
      batchState.d_output_diff_, 1,
      d_wo_, N));
  }

  // Load data from dict while gpu is running async
  batchState.preLoadLine();

  hipStreamSynchronize(batchState.stream_);

  // Copy d_lossValue_ -> lossValue, d_grad_ -> grad
  if( backprop ) {
    CUDA_CHECK(hipMemcpy(grad.data(), batchState.d_grad_, N*sizeof(real), hipMemcpyDeviceToHost));
  }
  CUDA_CHECK(hipMemcpy(&lossValue, batchState.d_lossValue_, sizeof(real), hipMemcpyDeviceToHost));
}

} // namespace fasttext
